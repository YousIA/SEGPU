#include <iostream>
#include <hip/hip_runtime.h>

//#include <time.h>
//#include <cutil.h> 

using namespace std;
# define r 40
# define M 1000  // number of transactions
# define N 40 // number of items
# define alpha 1 // represents the weight of the support in the first fitness function
# define Beta 1 // represents the weight of the confidence in the first fitness function
struct ligne {int trans[N]; int nb;} *lg;
struct bee {int solution[N]; float cost; float lift; float leverage; float coverage;} *be;
/**************prototype declaration*******/
void read_trans(ligne T[]);// this function allows to read the transactional data base et insert it into the dataset vector
void display_dataset(ligne T[]); //this function allows to display the transactional data base
void display_solution(bee S); // this function display the current solution with its cost
float support_rule(ligne T[], int s[]); // this function calculates the support of the entire solution s
float support_antecedent(ligne T[], int s[]); // this function computes the support of the antecedent of the solution s
float confidence(int s[]); // it calculates the confidence of the rule
float confidence(int sr, int sa); // it calculates the confidence of the rule
float fitness1(int sr, int sa); // computes the fitness of a given solution s
void create_Sref(bee *s, ligne V[]); // here we create the solution reference sref and initialize it with the random way
bee neighborhood_computation(bee S, ligne V[], int k);// this function explores the local region for each bee
void search_area1(bee s, bee *T, int iteration, ligne V [], int k, int flip); //detremines the search area for each bee using the first strategy
void search_area2(bee s, bee *T, int iteration, ligne V[], int k, int flip); //detremines the search area for each bee using the second strategy
void search_area3(bee s, bee *T, int iteration, ligne V[], int k, int distance); //detremines the search area for each bee using the third strategy
int W(int t[]); // indicates the  weight of solution representing by a vector t, this function is used on search_area3()
void copy(int t[], int v[]); // it copies the vector t in the vector v
int best_dance(bee *T, int k); // return the best dance after the exploration of search region of each bee
void parallel_fitness(bee *solution, ligne V[]); // parallelize solution computing 
void display_bees(bee T[], int k); // display solutions
/*************************************************************************************/
__global__ void KernelSupport_rules(int *s_GPU, int *compt_GPU, struct ligne *dataset_GPU){
	int thread_idx ;
	thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
	bool appartient=true;
        int Td1=0;
        int Td2=0;
      // for (int i=0; i<2;i++)
       //{
         if (thread_idx <M){ 
		int j=1;
		while (j<N){
			if (s_GPU[j]!=0){
				int l=0; 
				bool existe=false;
				while (l< dataset_GPU[thread_idx/*(2*thread_idx)+i*/].nb && existe==false){
					if (dataset_GPU[thread_idx/*(2*thread_idx)+i*/].trans[l]==j){
						existe=true;
					}
					l++;
				}
				if (existe==false){
					appartient=false;
				}
			}
			j++;    
		}
		if (appartient==true){
			compt_GPU[thread_idx/*(2*thread_idx)+i*/]=1;
		}
	}
   // }
}

__global__ void KernelSupport_antecedent(int *s_GPU, int *compt_GPU, struct ligne *dataset_GPU){
	int thread_idx ;
	thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
	bool appartient=true;
         if (thread_idx <M){
          //for (int i=0;i<2;i++)
            //{ 
		int j=1;
		while (j<N){
			if (s_GPU[j]==1){
				int l=0; 
				bool existe=false;
				while (l< dataset_GPU[thread_idx/*(2*thread_idx)+i*/].nb && existe==false){
					if (dataset_GPU[thread_idx/*(2*thread_idx)+i*/].trans[l]==j){
						existe=true;
					}
					l++;
				}
				if (existe==false){
					appartient=false;
				}
			}
			j++;    
		}
		if (appartient==true){
			compt_GPU[thread_idx/*(2*thread_idx)+i*/]=1;
		}
	  }
       //}
}
__global__ void KernelSupport_consequent(int *s_GPU, int *compt_GPU, struct ligne *dataset_GPU){
	int thread_idx ;
	thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
	bool appartient=true;
         if (thread_idx <M){
          //for (int i=0;i<2;i++)
            //{ 
		int j=1;
		while (j<N){
			if (s_GPU[j]==2){
				int l=0; 
				bool existe=false;
				while (l< dataset_GPU[thread_idx/*(2*thread_idx)+i*/].nb && existe==false){
					if (dataset_GPU[thread_idx/*(2*thread_idx)+i*/].trans[l]==j){
						existe=true;
					}
					l++;
				}
				if (existe==false){
					appartient=false;
				}
			}
			j++;    
		}
		if (appartient==true){
			compt_GPU[thread_idx/*(2*thread_idx)+i*/]=1;
		}
	  }
       //}
}

int main(void){
    FILE *f=NULL;
    f=fopen("/home/ydjenouri/mesprog/resultat1.txt","a");
    struct ligne *dataset_CPU, *dataset_GPU;
    struct bee *T_Dance;
    struct bee Sref;
    struct bee best;
    int k=5, flip=1, distance, IMAX=1;
    hipEvent_t start, stop;
    float  elapsedTime;
    int j;
    /*****************************parallel program***********************/
    
  //for ( k=5; k<=15;k=k+5)
   //{
    //for ( flip=1; flip<=(N/5); flip++)
     //{
     //for (IMAX=1; IMAX<= 20;IMAX++)
     //{
  
    
	//allocation de la memoire dans le CPU
	dataset_CPU = (ligne *) malloc(M * sizeof(ligne)) ;
	T_Dance = (bee *) malloc(k * sizeof(bee)) ;
       ////allocation de la memoire dans le GPU
       hipMalloc( (void**) &dataset_GPU, M*sizeof(ligne));

       read_trans(dataset_CPU); // read transactional database and insert in the dataset_CPU
	hipMemcpy(dataset_GPU, dataset_CPU, M * sizeof(ligne), hipMemcpyHostToDevice);
      	  hipEventCreate( &start );
         hipEventCreate( &stop );
         hipEventRecord( start, 0 ) ;
       create_Sref(&Sref, dataset_GPU); // creer une solution reference
	//display_solution(Sref);
      search_area1(Sref, T_Dance, IMAX, dataset_GPU, k,flip);
     // display_bees(T_Dance);
      for ( int i=0; i<IMAX;i++)
	{
	    			
	    for ( j=0;j<k;j++) // neighborhood computation for all the solution in tab
					{ 
				       T_Dance[j]=neighborhood_computation(T_Dance[j], dataset_GPU,k);
					}
					j=best_dance(T_Dance,k);
					copy(T_Dance[j].solution,Sref.solution);
					Sref.cost=T_Dance[j].cost;
					if (Sref.cost > best.cost)//atte o maximisation
			    { 
					 copy(Sref.solution, best.solution);
					 best.cost=Sref.cost;
			    }
		       display_bees(T_Dance,k);	 
//			//average=best.cost+average; 	        
			//printf("\nk="+b.k+" IMAX="+b.IMAX+"  average fitness="+average);
                     search_area1(Sref,T_Dance,i, dataset_GPU,k,flip);
	
       } //Bso ending

    hipEventRecord( stop, 0 ) ;
    hipEventSynchronize( stop ) ;
    hipEventElapsedTime( &elapsedTime,start, stop ) ;
    printf("K=%d IMAX=%d  Execution Time in GPU : %3.1f ms\n", k,IMAX, elapsedTime );
    //parallel_fitness(&T_Dance[0], dataset_GPU);
    //printf ("cost=%f\n", T_Dance[0].cost);  
    //printf ("lift=%f  leverage=%f coverage= %f\n", best.lift, best.leverage,best.coverage);
   /// fprintf(f,"K=%d IMAX=%d flip=%d Execution Time in GPU : %3.1f ms\n", k,flip,IMAX, elapsedTime );
    printf("Yes\n");
    hipEventDestroy( start );
    hipEventDestroy( stop );
//}// end loop IMAX
//} // end loop flip
//} // end loop k

//fclose(f);
hipFree(dataset_GPU);

return 0;
}
/**********************copry t in v********/
void copy(int t[], int v[])
{
for (int i=0;i<N; i++)
{
v[i]=t[i]; 
}     
}
/*******read transactional data bass and insert it in the data set structure********************************/
void read_trans(ligne T[]){
	char c='4';
	char t[100];
	int j;
	int i=0;
	int l=0;
	FILE *f=NULL;
	f=fopen("/home/ydjenouri/mesprog/dataset.txt","r");
	if (f!=NULL) {
		//cout<<"the file is succefully opened"<<endl;
		j=0;
		while (c!=EOF){
			c=fgetc(f);
			if (c==' '){
				t[j]='\0';
				T[i].trans[l]=atoi(t);
                            l++;
				j=0;
			}
			if (c=='\n'){
				T[i].nb=l;
				l=0;
				i++;
				j=0;
			}
			if (c!=' ' && c!='\n'){
				t[j]=c;
				j++;
			}
		}   
		fclose(f);
	}
}
/*************************compute the support of the solution s**********/
float support_rule(ligne T[], int s[])
{
float compt=0;
		for (int i=0; i<M; i++)
		{
		bool appartient=true;
		
		int j=1;
		while (j<N)
		{
		 if (s[j]!=0)
		{
			int l=0; 
			bool existe=false;
			while (l< T[i].nb && existe==false)
			{
				if (T[i].trans[l]==j)
				{existe=true;}
			l++;
			}
			if (existe==false){appartient=false;}
		}
		j++;	
		}
		if (appartient==true) {compt++;}
		}
	   compt=compt/M;
	return compt;
}
/*****************************support antecedent computing*****************************/
float support_antecedent(ligne T[], int s[])
	{
             float compt=0;
		
		for (int i=0; i<M; i++)
		{
		bool appartient=true;
		int j=1;
		while (j<N)
		{
		 if (s[j]==1 ||s[j]==2)
		{
			int l=0; 
			bool existe=false;
			while (l< T[i].nb && existe==false)
			{
					if (T[i].trans[l]==j)
				        {existe=true;}
			l++;
			}
			if (existe==false){appartient=false;}
		}
		j++;	
		}
		if (appartient==true) {compt++;}
		}
	   compt=compt/M;
	//if(compt!=0)System.out.println("antecedent"+compt);
	   return compt;
	}
/****************************condifence computing**************************/
float confidence(int sr, int sa)
{
	float conf=1;
	conf=(float)sr/sa;
return conf;
}
/***********************evaluation of the solution s******/
float fitness1(int sr, int sa)
	{
	float cost=0; 
	//if (support_rule(sol)<Minsup || confidence(sol)<Minconf){cout=-1;}
	float x=(float)alpha*(sr/M);
	float y=(float)Beta*confidence(sr,sa);
	cost=x+y;
	return cost;
	}
/**************************display_solution*****************/
void display_solution(bee S)
{
for (int i=0;i<N;i++)
{
    printf("%d ", S.solution[i]);
}
printf ("cost is:%f",S.cost);
printf("\n");
}

/*********************create a solution reference Sref******************************************/
void create_Sref(bee *s, ligne V[])
{
	for (int i=0;i<N;i++){
		if (rand() % 2==0){
			(*s).solution[i]=0 ;
		}
		else {
			if (rand() % 2==0){
				(*s).solution[i]=0;
			}
			else {
				(*s).solution[i]=rand() % 3; 
			}
		}
	}
       parallel_fitness(s, V);
}
/***********************************negihborhood computation************************/
bee neighborhood_computation(bee S, ligne V[], int k)
{
bee s;
int indice=0;
int i=0; 
bee neighbor, best_neighbor;
float best_cost=0;
		//copy(S.solution,best_neighbor);
		   copy(S.solution,neighbor.solution);
                  while (i<k)
		   {
	          	    
	          if (neighbor.solution[indice]==0) 
	          {
	        		  if (rand()%2==0)
	        		  {neighbor.solution[indice]=1;}
	        		  else{neighbor.solution[indice]=2;}
	          }
	          else {
	          if (neighbor.solution[indice]==1) 
	          {
	        	  if (rand()%2==0)
                         neighbor.solution[indice]=0;

	        	  else {
	        	 neighbor.solution[indice]=2;
	        		  }
	          }
	          else {
	          if (neighbor.solution[indice]==2) 
	          {
	        	  	  if (rand()%2==0)
                                neighbor.solution[indice]=0;

	        	  else {
	                neighbor.solution[indice]=1;
	        		 }
	          }
	          }
	          }
	     indice++;
	     if (indice>=N){indice=0;}   
	

               parallel_fitness(&neighbor, V); 
		 if (neighbor.cost>best_cost){copy(neighbor.solution,best_neighbor.solution);
                                            best_cost=neighbor.cost;}
		 i++;
		 }
copy(best_neighbor.solution, s.solution);
s.cost=best_cost;
return s;
}
/************************determination of search area********************/
void search_area1(bee s, bee *T, int iteration, ligne V[], int k, int flip)
{
	 
	    int indice=iteration % N;
	    int i=0;
		   while (i<k)
		   {
			   for (int j=0;j<N;j++)
			   {   
			    T[i].solution[j]=s.solution[j];	    
			   }	
	                 if (T[i].solution[indice]==0) 
	                {
	        	         if (iteration%4==0)
	        		  {T[i].solution[indice]=1;}
	        		  else{T[i].solution[indice]=2;}
	        		  
	        	    //           }    
	                }
			
	          else{
	          if (T[i].solution[indice]==1) 
	          {	  if (iteration%3==0)
			  {T[i].solution[indice]=0;}
			  else{T[i].solution[indice]=2;}
	        		 
	        }
	          else{ 
	          
	        	  if (iteration%2==0)
	    		  {
                       T[i].solution[indice]=1;}
	    		  else{
                         T[i].solution[indice]=0;}
	        	 }
	          }
	     indice=indice+flip;
	     if (indice>=N){indice=0;}   
		 parallel_fitness(&T[i], V);
		//T_Dance[i].cost=fitness1(T_Dance[i].solution);//evaluer solution  
		 i++;
		   }
}
/**************search 2*********************/
void search_area2(bee s, bee *T, int iteration, ligne V[], int k, int flip)
{
int i=0;
int Nb_sol=0;
bool stop=false;
	  while (i<N && stop==false)
	  {
		   for (int j=0;j<N;j++)
		   {   
			   T[Nb_sol].solution[j]=s.solution[j];	       
		   }
		   for (int l=i;l<(i+flip)%N;l++)
		   {
		  if ( T[Nb_sol].solution[l]==0) 
	     {
	   	  if (rand()%2==1)
	   		  { T[Nb_sol].solution[l]=1;}
	   		  else{T[Nb_sol].solution[l]=2;}
	   		  
	   	}
	     else {
	     if (T[Nb_sol].solution[l]==1) 
	     {
	    	 if (rand()%2==1)
	  		  {T[Nb_sol].solution[l]=0;}
	  		  else{T[Nb_sol].solution[l]=2;}
	     }
	     else {
	       if (T[Nb_sol].solution[l]==2) 
	        {
	    	 if (rand()%2==0)
	  		  {T[Nb_sol].solution[l]=0;}
	  		  else{T[Nb_sol].solution[l]=1;}
	         }
	         }
	     }
		}
         parallel_fitness(&T[i], V);
	//T_Dance[Nb_sol].cost=fitness1(T_Dance[Nb_sol].solution); //evaluates the solution  
	Nb_sol++; 
	if (Nb_sol==k){stop=true;}   
	}
}   
/********search3***************************/
int W(int t[])
{
int w=0;
	for (int i=0;i<N; i++)
	{
	w=w+t[i];
	}
return w;
} 
/*******search 3 continued****************************/
void search_area3(bee s, bee *T, int iteration, ligne V[], int k, int distance)
{
int Nb_sol=0;
	  while (Nb_sol!=k)
	  {
		   for (int j=0;j<N;j++)
		   {   
			  T[Nb_sol].solution[j]=s.solution[j];	 	    
		   }
		   int l=0;
		   int cpt=0;
		   while (cpt<distance)
		   {
		  if (T[Nb_sol].solution[l]==0) 
	     {
	   	  if (rand()%2==1)
	   		  {T[Nb_sol].solution[l]=1; cpt++;}
	   		  else{T[Nb_sol].solution[l]=2;cpt=cpt+2;}
	   		  
	   	}
	     else {
	     if (T[Nb_sol].solution[l]==1) 
	     {
	    	 if (rand()%2==0)
	  		  {T[Nb_sol].solution[l]=0;cpt++;}
	  		  else{T[Nb_sol].solution[l]=2;cpt++;}
	     }
	     else {
	       if (T[Nb_sol].solution[l]==2) 
	        {
	    	 if (rand()%2==0)
	  		  {T[Nb_sol].solution[l]=0;cpt=cpt+2;}
	  		  else{T[Nb_sol].solution[l]=1;cpt=cpt+1;}
	         }
	         }
	     
		   }
		  l=(l+1)%N;
		   } //end the small while
         parallel_fitness(&T[Nb_sol], V);
	//T_Dance[Nb_sol].cost=fitness1(T_Dance[Nb_sol].solution);//assecees the solution  
	Nb_sol++; 
	  } // end the big while
 }
/********************************best dance********************/
int best_dance(bee *T, int k)
{
	float max=T[0].cost;
	int indice=0;
	for (int i=1;i<k;i++)	
	{
     	if (T[i].cost>max)
	     {     
           max=T[i].cost;
		   indice=i;
         }
	}
return indice;
}
/***********************paralelize solution computing*******/
void parallel_fitness(bee *sol, ligne V[])
{
       int *s_GPU;
	int *compt;
       int *s_CPU;
       s_CPU = (int *) malloc(N*sizeof(int));
       for (int i=0;i<N;i++)
       {
        s_CPU[i]=(*sol).solution[i]; 
       }
      	compt = (int *) malloc(M*sizeof(int));
       	for (int i=0;i<M;i++){
		compt[i]=0;
	}
	
       int *compt_GPU;
	// cudaEventCreate( &start );
     	// cudaEventCreate( &stop );
     	// cudaEventRecord( start, 0 ) ;

	hipMalloc( (void**) &s_GPU, N*sizeof(int));
	hipMalloc( (void**) &compt_GPU, M* sizeof(int));
	hipMemcpy(s_GPU, s_CPU, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(compt_GPU, compt, M *sizeof(int),hipMemcpyHostToDevice);

	KernelSupport_rules<<<1000,41>>>(s_GPU, compt_GPU, V);
       hipMemcpy(compt, compt_GPU, M*sizeof(int),hipMemcpyDeviceToHost);
       int sr=0; 
       for (int i=0;i<M;i++){
        	sr=sr+compt[i];
       }
       KernelSupport_antecedent<<<1000,41>>>(s_GPU, compt_GPU, V);
       hipMemcpy(compt, compt_GPU, M*sizeof(int),hipMemcpyDeviceToHost);
       int sa=0; 
       for (int i=0;i<M;i++){
        	sa=sa+compt[i];
       }
       KernelSupport_consequent<<<1000,41>>>(s_GPU, compt_GPU, V);
       hipMemcpy(compt, compt_GPU, M*sizeof(int),hipMemcpyDeviceToHost);
       int sc=0; 
       for (int i=0;i<M;i++){
        	sc=sc+compt[i];
       }

       (*sol).cost=fitness1(sr,sa);
       float conf=1;
	conf=(float)sr/sa;
       (*sol).lift=conf/sc;
       (*sol).leverage=sr-(sc*sa);
       (*sol).coverage=sc;
}
/*****************************display T_dance************/
void display_bees(bee T[], int k)
{
//FILE *f=NULL;
//f=fopen("/home/ydjenouri/mesprog/resultat1.txt","a");
//if (f!=NULL) {
for (int i=0;i<k;i++)
{
    for (int j=0;j<N;j++)
    {
    printf ("%d ",T[i].solution[j]);   
    }
    printf("%f", T[i].cost);
    printf("\n");
}
//fclose(f);
//}
}
